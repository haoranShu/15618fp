#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <math.h>

#include "heatmap.h"

__global__ void KDE_renderer_kernel(float* hm, unsigned w, unsigned h,
                                    float* xs, float* ys, float* ws, unsigned num_points,
                                    float x_min, float x_max, float y_min, float y_max,
                                    float KDE_sd)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_ths = blockDim.x * gridDim.x;

    float x_norm, y_norm;
    float x_range = x_max - x_min;
    float y_range = y_max - y_min;
    float exp_scalar = - 1 / (2 * KDE_sd * KDE_sd);
    float scalar = 1 / (KDE_sd * sqrt(2 * M_PI));
    float hx = idx % w;
    float hy = idx / w;
    for (int i = idx; i < w * h; i += total_ths) {
        for (int j = 0; j < num_points; j++) {
            x_norm = (xs[j] - x_min) / x_range * w - hx;
            y_norm = (ys[j] - y_min) / y_range * h - hy;
            hm[i] += ws[j] * exp((x_norm * x_norm + y_norm * y_norm) * exp_scalar);
        }
        hm[i] *= scalar;
    }
}

void cudaKDE_renderer(heatmap_t* h, float* xs, float* ys, float* ws, unsigned num_points,
                      float x_min, float x_max, float y_min, float y_max, float KDE_sd)
{
    float* cudaH_buf;
    float* cuda_xs;
    float* cuda_ys;
    float* cuda_ws;

    hipMalloc(&cudaH_buf, sizeof(float) * h->w * h->h);
    hipMalloc(&cuda_xs, sizeof(float) * num_points);
    hipMalloc(&cuda_ys, sizeof(float) * num_points);
    hipMalloc(&cuda_ws, sizeof(float) * num_points);

    hipMemset(cudaH_buf, 0, sizeof(float) * h->w * h->h);
    hipMemcpy(cuda_xs, xs, sizeof(float) * num_points, hipMemcpyHostToDevice);
    hipMemcpy(cuda_ys, ys, sizeof(float) * num_points, hipMemcpyHostToDevice);
    hipMemcpy(cuda_ws, ws, sizeof(float) * num_points, hipMemcpyHostToDevice);

    KDE_renderer_kernel<<<1024, 1024>>>(cudaH_buf, h->w, h->h, cuda_xs, cuda_ys, cuda_ws,
                                      num_points, x_min, x_max, y_min, y_max, KDE_sd);

    hipMemcpy(h->buf, cudaH_buf, sizeof(float) * h->w * h->h, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}
