#include "hip/hip_runtime.h"
#include <string>
#include <ctime>
#include <iostream>

#include "gl_utility.h"

clock_t start;

__global__ void renderNewPointsKernel(float x0, float y0, float w, float h, 
    int W, int H, float* buf, Quadtree_node* nodes, float pt_width, float pt_width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < W * H; i += blockDim.x * gridDim.x) {
        buf[i] = 0;
        float pt_x = x0 + (i%W + 0.5) * w / W;
        float pt_y = y0 + (i/W + 0.5) * h / H;
        traverse(nodes, buf+i, pt_width, pt_height, pt_x, pt_y);
    }
}

__global__ void writeToImageKernel(float* weights, float* color, int num_pixels)
{

}

void renderNewPointsCUDA(float x0, float y0, float w, float h, std::string filename)
{
    start = std::clock();
    float pt_width = w * 9 / renderW;
    float pt_height = h * 9 / renderH;
    renderNewPointsKernel<<<128, 128>>>(x0, y0, w, h, renderW, renderH,
        pixel_weights, nodes, pt_width, pt_height);
    writeToImageKernel<<<128, 128>>>(pixel_weights, pixel_color, renderH * renderW);

    std::cout << (std::clock() - start) * 1000  / (double) CLOCKS_PER_SEC << " ms\n";
    hipMemcpy((void *)ppmOutput->data, (void *)pixel_color,
        renderH * renderW * sizeof(unsigned char), hipMemcpyDeviceToHost);
    writePPMImage(ppmOutput, filename);
}

void setupTextureCUDA()
{

}

void renderSceneCUDA()
{
    
}

void zoomingCUDA(int button, int state, int x, int y)
{

}