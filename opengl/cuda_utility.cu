#include "hip/hip_runtime.h"
#include <string>
#include <ctime>
#include <iostream>

#include "gl_utility.h"

clock_t start;

__global__ renderNewPointsKernel(float x0, float y0, float w, float h, 
    int W, int H, float* buf, Quadtree_node* nodes, float pt_width, float pt_width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < W * H; i += blockDim.x * gridDim.x) {
        float pt_x = x0 + (i%W + 0.5) * w / W;
        float pt_y = y0 + (i/W + 0.5) * h / H;
        traverse(nodes, buf+i, pt_width, pt_height, pt_x, pt_y);
    }
}

void renderNewPointsCUDA(float x0, float y0, float w, float h, std::string filename)
{
    start = std::clock();
    float pt_width = w * 9 / renderW;
    float pt_height = h * 9 / renderH;
    renderNewPointsKernel<<<128, 128>>>(x0, y0, w, h, renderW, renderH,
        cudabuf, nodes, pt_width, pt_height);
    writeToImageKernel<<<128, 128>>>;

    std::cout << (std::clock() - start) * 1000  / (double) CLOCKS_PER_SEC << " ms\n";
    hipMemcpy();
    writePPMImage(ppmOutput, filename);
}

void setupTextureCUDA()
{

}

void renderSceneCUDA()
{
    
}

void zoomingCUDA(int button, int state, int x, int y)
{

}