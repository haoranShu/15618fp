#include "hip/hip_runtime.h"
#include <string>
#include <ctime>
#include <iostream>

#include "gl_utility.h"

clock_t start;

__global__ void renderNewPointsKernel(float x0, float y0, float w, float h, 
    int W, int H, float* buf, Quadtree_node* nodes, Points *pts, float pt_width, float pt_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < W * H; i += blockDim.x * gridDim.x) {
        buf[i] = 0;
        float pt_x = x0 + (i%W + 0.5) * w / W;
        float pt_y = y0 + (i/W + 0.5) * h / H;

        Bounding_box box();
        region.set(pt_x, pt_y, pt_x + pt_width, pt_y + pt_height);
        Parameters params(12, 64);
        traverse(nodes, 0, &buf[i], region, pts, params);
    }
}

__device__ 
void traverse(Quadtree_node *nodes, int idx, float *buf, Bounding_box &box, 
        Points *pts, Parameters params)
{
    Quadtree_node current = nodes[idx];
    if (!box.overlaps(current.bounding_box()))
        return;

    if (box.contains(current.bounding_box())) 
    {
         *buf = *buf + current.num_points();
         return;
    }

    if (params.depth == params.max_depth || current.num_points() <= params.min_points_per_node)
    {
        for (int it = node.points_begin() ; it < node.points_end() ; ++it)
        {
            float2 p = pts->get_point(it);
            if (!box.contains(p))
                *buf = *buf + 1;
        }
        return;
    }
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+0, buf, box, pts, Parameters(params, true));
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+1, buf, box, pts, Parameters(params, true));
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+2, buf, box, pts, Parameters(params, true));
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+3, buf, box, pts, Parameters(params, true));
}

__global__ void writeToImageKernel(float* weights, float* color, int num_pixels)
{

}

void renderNewPointsCUDA(float x0, float y0, float w, float h, std::string filename)
{
    start = std::clock();
    float pt_width = w * 9 / renderW;
    float pt_height = h * 9 / renderH;
    renderNewPointsKernel<<<128, 128>>>(x0, y0, w, h, renderW, renderH,
        pixel_weights, nodes, pt_width, pt_height);
    writeToImageKernel<<<128, 128>>>(pixel_weights, pixel_color, renderH * renderW);

    std::cout << (std::clock() - start) * 1000  / (double) CLOCKS_PER_SEC << " ms\n";
    hipMemcpy((void *)ppmOutput->data, (void *)pixel_color,
        renderH * renderW * sizeof(unsigned char), hipMemcpyDeviceToHost);
    writePPMImage(ppmOutput, filename);
}

void setupTextureCUDA()
{

}

void renderSceneCUDA()
{
    
}

void zoomingCUDA(int button, int state, int x, int y)
{

}
