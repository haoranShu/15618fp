#include "hip/hip_runtime.h"
#include <string>
#include <ctime>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_renderer.h"
#include "cdpQuadtree.h"

clock_t start_cuda;

__device__ void traverse(Quadtree_node* nodes, float* weight,
    float pt_width, float pt_height, float pt_x, float pt_y)
{

}

__global__ void renderNewPointsKernel(float x0, float y0, float w, float h, 
    int W, int H, float* buf, Quadtree_node* nodes, float pt_width, float pt_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < W * H; i += blockDim.x * gridDim.x) {
        buf[i] = 0;
        float pt_x = x0 + (i%W + 0.5) * w / W;
        float pt_y = y0 + (i/W + 0.5) * h / H;
        traverse(nodes, buf+i, pt_width, pt_height, pt_x, pt_y);
    }
}

template <int blockSize>
__global__ void reduceMaxKernel(float* src, float* dst, int n)
{
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * (blockSize * 2) + tid;
    int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;

    while (i < n) {
        sdata[tid] = src[i] > src[i + blockSize] ? src[i] : src[i + blockSize];
        i += gridSize;
    }
    __syncthreads();

    int startSize = 512;
    while (startSize > warpSize) {
        if (blockSize > startSize) {
            if (tid < startSize/2) { sdata[tid] = src[tid] > src[tid + startSize/2] ? src[tid] : src[tid + startSize/2]; }
            __syncthreads();
        }
        startSize /= 2;
    }

    // assuming a warpSize of 32
    if (tid < 32) {
        if (blockSize >= 64) {
            sdata[tid] = src[tid] > src[tid + 32] ? src[tid] : src[tid + 32];
        }
        if (blockSize >= 32) {
            sdata[tid] = src[tid] > src[tid + 16] ? src[tid] : src[tid + 16];
        }
        if (blockSize >= 16) {
            sdata[tid] = src[tid] > src[tid + 8] ? src[tid] : src[tid + 8];
        }
        if (blockSize >= 8) {
            sdata[tid] = src[tid] > src[tid + 4] ? src[tid] : src[tid + 4];
        }
        if (blockSize >= 4) {
            sdata[tid] = src[tid] > src[tid + 2] ? src[tid] : src[tid + 2];
        }
        if (blockSize >= 2) {
            sdata[tid] = src[tid] > src[tid + 1] ? src[tid] : src[tid + 1];
        }
    }

    if (tid == 0) dst[blockIdx.x] = sdata[0];
}

__global__ void writeToImageKernel(float* weights, unsigned char* color, int num_pixels,
    int max_weight, const heatmap_colorscheme_t* colorscheme)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    for (int i = idx; i < num_pixels; i += blockDim.x * gridDim.x) {
        float val = weights[i] / (float)max_weight;
        size_t color_idx = (size_t)((float)(colorscheme->ncolors-1)*val + 0.5f);
        color[4*i] = (colorscheme->colors)[color_idx*4];
        color[4*i+1] = (colorscheme->colors)[color_idx*4+1];
        color[4*i+2] = (colorscheme->colors)[color_idx*4+2];
        color[4*i+3] = (colorscheme->colors)[color_idx*4+3];
    }
}

void cudaInit()
{
    hipMalloc(&pixel_weights, renderH * renderW * sizeof(float));
    hipMalloc(&pixel_color, renderH * renderW * sizeof(unsigned char));
    //hipMalloc(&max_buf, 1 * sizeof(float));
    hipMalloc(&sizes, 2 * sizeof(int))

    hipMemcpy((void *)pixel_weights, (void *)hm->buf,
        renderH * renderW * sizeof(float), hipMemcpyHostToDevice);
}

__global__ void tempMax(float* src, float* dst, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    float &max_weight = dst[0];
    if (idx == 0) {
        for (int i = 0; i < n; i++) {
            max_weight = max_weight > src[i] ? max_weight : src[i];
        }
    }
}

void shrink(int n, int* sizes)
{
    int &g = sizes[0];
    int &b = sizes[1];
    if (n <= 2 * b) {
        g = 1;
        while (b > n) b >>= 1;
    } else {
        int m = (n + (b - 1)) / b;
        while (g > m) g >>= 1;
    }
}

void renderNewPointsCUDA(float x0, float y0, float w, float h, std::string filename,
    int* sizes)
{
    start_cuda = std::clock();
    float pt_width = w * 9 / renderW;
    float pt_height = h * 9 / renderH;
    //renderNewPointsKernel<<<128, 128>>>(x0, y0, w, h, renderW, renderH,
    //    pixel_weights, nodes, pt_width, pt_height);

    // get the maximum value of all weigths
    float max_weight;
    //tempMax<<<1, 1>>>(pixel_weights, max_buf, renderH * renderW);
    //hipMemcpy((void *)&max_weight, (void *)max_buf, 1 * sizeof(float), hipMemcpyDeviceToHost);
    
    int npixel = renderH * renderW;
    shrink(npixel, sizes);
    hipMalloc(&max_buf, (sizes[0] + sizes[0] >> 1) * sizeof(float));

    int slen = sizes[0];
    float* ps = pixel_weights;
    int smemSize = 0;
    if (slen > 1) {
        float* pd = max_buf + sizes[0];
        do {
            shrink(slen, sizes);
            smemSize = sizes[1] * sizeof(float);
            reduceMaxKernel<<<sizes[0], sizes[1], smemSize>>>(ps, pd, sizes[0]);
            float *pt = ps;
            ps = pd;
            pd = pt;
        } while (slen > 1);
    }

    max_weight = ps[0];

    hipDeviceSynchronize();
    start_cuda = std::clock();
    writeToImageKernel<<<128, 128>>>(pixel_weights, pixel_color, npixel, max_weight, heatmap_cs_default);
    hipDeviceSynchronize();
    std::cout << (std::clock() - start_cuda) * 1000  / (double) CLOCKS_PER_SEC << " ms\n";
    hipMemcpy((void *)ppmOutput->data, (void *)pixel_color,
        npixel * sizeof(unsigned char), hipMemcpyDeviceToHost);
    writePPMImage(ppmOutput, filename);
}