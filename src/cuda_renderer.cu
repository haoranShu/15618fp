#include "hip/hip_runtime.h"
#include <string>
#include <ctime>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_renderer.h"
#include "cdpQuadtree.h"

clock_t start_cuda;

__device__ void traverse(Quadtree_node* nodes, float* weight,
    float pt_width, float pt_height, float pt_x, float pt_y)
{

}

__global__ void renderNewPointsKernel(float x0, float y0, float w, float h, 
    int W, int H, float* buf, Quadtree_node* nodes, float pt_width, float pt_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < W * H; i += blockDim.x * gridDim.x) {
        buf[i] = 0;
        float pt_x = x0 + (i%W + 0.5) * w / W;
        float pt_y = y0 + (i/W + 0.5) * h / H;
        traverse(nodes, buf+i, pt_width, pt_height, pt_x, pt_y);
    }
}

__global__ void reduceMaxKernel(float* src, float* dst, int n)
{
    extern __shared__ float sdata[];

    int blockSize = blockDim.x;
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockSize * 2) + tid;
    int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;
    float temp = 0;

    while (i < n - blockSize) {
        temp = src[i] > src[i + blockSize] ? src[i] : src[i + blockSize];
        sdata[tid] = sdata[tid] > temp ? sdata[tid] : temp;
        i += gridSize;
    }
    while (i < n) {
        sdata[tid] = sdata[tid] > src[i] ? sdata[tid] : src[i]; 
    }
    __syncthreads();

    int startSize = 512;
    while (startSize > warpSize) {
        if (blockSize > startSize) {
            if (tid < startSize/2) { sdata[tid] = sdata[tid] > sdata[tid + startSize/2] ? sdata[tid] : sdata[tid + startSize/2]; }
            __syncthreads();
        }
        startSize /= 2;
    }

    // assuming a warpSize of 32
    if (tid < 32) {
        if (blockSize >= 64) {
            sdata[tid] = sdata[tid] > sdata[tid + 32] ? sdata[tid] : sdata[tid + 32];
        }
        if (blockSize >= 32) {
            sdata[tid] = sdata[tid] > sdata[tid + 16] ? sdata[tid] : sdata[tid + 16];
        }
        if (blockSize >= 16) {
            sdata[tid] = sdata[tid] > sdata[tid + 8] ? sdata[tid] : sdata[tid + 8];
        }
        if (blockSize >= 8) {
            sdata[tid] = sdata[tid] > sdata[tid + 4] ? sdata[tid] : sdata[tid + 4];
        }
        if (blockSize >= 4) {
            sdata[tid] = sdata[tid] > sdata[tid + 2] ? sdata[tid] : sdata[tid + 2];
        }
        if (blockSize >= 2) {
            sdata[tid] = sdata[tid] > sdata[tid + 1] ? sdata[tid] : sdata[tid + 1];
        }
    }

    if (tid == 0) dst[blockIdx.x] = sdata[0];

    if (tid == 0) printf("returned %d\n", blockIdx.x);
    return;
}

__global__ void writeToImageKernel(float* weights, unsigned char* color, int num_pixels,
    int max_weight, const heatmap_colorscheme_t* colorscheme)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    for (int i = idx; i < num_pixels; i += blockDim.x * gridDim.x) {
        float val = weights[i] / (float)max_weight;
        size_t color_idx = (size_t)((float)(colorscheme->ncolors-1)*val + 0.5f);
        color[4*i] = (colorscheme->colors)[color_idx*4];
        color[4*i+1] = (colorscheme->colors)[color_idx*4+1];
        color[4*i+2] = (colorscheme->colors)[color_idx*4+2];
        color[4*i+3] = (colorscheme->colors)[color_idx*4+3];
    }
}

void cudaInit()
{
    hipMalloc(&pixel_weights, renderH * renderW * sizeof(float));
    hipMalloc(&pixel_color, renderH * renderW * sizeof(unsigned char));
    //hipMalloc(&max_buf, 1 * sizeof(float));
    //hipMalloc(&sizes, 2 * sizeof(int));

    hipMemcpy((void *)pixel_weights, (void *)hm->buf,
        renderH * renderW * sizeof(float), hipMemcpyHostToDevice);
}

__global__ void tempMax(float* src, float* dst, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    float &max_weight = dst[0];
    if (idx == 0) {
        for (int i = 0; i < n; i++) {
            max_weight = max_weight > src[i] ? max_weight : src[i];
        }
    }
}

void shrink(int n, int* sizes)
{
    if (n <= 2 * sizes[1]) {
        sizes[0] = 1;
        while (sizes[1] > n) sizes[1] >>= 1;
    } else {
        int m = (n + (sizes[1] - 1)) / sizes[1];
        while (sizes[0] > m) sizes[0] >>= 1;
    }
}

void renderNewPointsCUDA(float x0, float y0, float w, float h, std::string filename,
    int* sizes)
{
    start_cuda = std::clock();
    float pt_width = w * 9 / renderW;
    float pt_height = h * 9 / renderH;
    //renderNewPointsKernel<<<128, 128>>>(x0, y0, w, h, renderW, renderH,
    //    pixel_weights, nodes, pt_width, pt_height);

    // get the maximum value of all weigths
    float max_weight;
    //tempMax<<<1, 1>>>(pixel_weights, max_buf, renderH * renderW);
    //hipMemcpy((void *)&max_weight, (void *)max_buf, 1 * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("here\n");
    sizes[0] = 512;
    sizes[1] = 256;
    int npixel = renderH * renderW;
    shrink(npixel, sizes);
    printf("here\n");
    hipMalloc(&max_buf, (sizes[0] + sizes[0] >> 1) * sizeof(float));

    int slen = sizes[0];
    float* ps = pixel_weights;
    int smemSize = 0;
    int blockSize;
    printf("here\n");
    if (slen > 1) {
        float* pd = max_buf + sizes[0];
        do {
            printf("%d %d\n", sizes[0], sizes[1]);
            shrink(slen, sizes);
            smemSize = sizes[1] * sizeof(float);
            blockSize = sizes[1];
            reduceMaxKernel<<<sizes[0], sizes[1], smemSize>>>(ps, pd, sizes[0]);
            float *pt = ps;
            ps = pd;
            pd = pt;
            slen = sizes[0];
        } while (slen > 1);
    }
    printf("here\n");

    hipMemcpy((void *)&max_weight, (void *)ps, 1 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    start_cuda = std::clock();
    writeToImageKernel<<<128, 128>>>(pixel_weights, pixel_color, npixel, max_weight, heatmap_cs_default);
    hipDeviceSynchronize();
    std::cout << (std::clock() - start_cuda) * 1000  / (double) CLOCKS_PER_SEC << " ms\n";
    hipMemcpy((void *)ppmOutput->data, (void *)pixel_color,
        npixel * sizeof(unsigned char), hipMemcpyDeviceToHost);
    writePPMImage(ppmOutput, filename);
}