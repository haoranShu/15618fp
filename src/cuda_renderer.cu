#include "hip/hip_runtime.h"
#include <string>
#include <ctime>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_renderer.h"
#include "cdpQuadtree.h"

clock_t start_cuda;

__device__ void traverse(Quadtree_node* nodes, float* weight,
    float pt_width, float pt_height, float pt_x, float pt_y)
{

}

__global__ void renderNewPointsKernel(float x0, float y0, float w, float h, 
    int W, int H, float* buf, Quadtree_node* nodes, float pt_width, float pt_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < W * H; i += blockDim.x * gridDim.x) {
        buf[i] = 0;
        float pt_x = x0 + (i%W + 0.5) * w / W;
        float pt_y = y0 + (i/W + 0.5) * h / H;
        traverse(nodes, buf+i, pt_width, pt_height, pt_x, pt_y);
    }
}

__global__ void reduceMaxKernel(float* src, float* dst, int n)
{
    extern __shared__ float sdata[];

    int blockSize = blockDim.x;
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockSize * 2) + tid;
    int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;
    float temp = 0;

    while (i < n - blockSize) {
        temp = src[i] > src[i + blockSize] ? src[i] : src[i + blockSize];
        sdata[tid] = sdata[tid] > temp ? sdata[tid] : temp;
        i += gridSize;
    }
    while (i < n) {
        sdata[tid] = sdata[tid] > src[i] ? sdata[tid] : src[i]; 
    }
    __syncthreads();

    int startSize = 512;
    while (startSize > warpSize) {
        if (blockSize > startSize) {
            if (tid < startSize/2) { sdata[tid] = sdata[tid] > sdata[tid + startSize/2] ? sdata[tid] : sdata[tid + startSize/2]; }
            __syncthreads();
        }
        startSize /= 2;
    }

    // assuming a warpSize of 32
    if (tid < 32) {
        if (blockSize >= 64) {
            sdata[tid] = sdata[tid] > sdata[tid + 32] ? sdata[tid] : sdata[tid + 32];
        }
        if (blockSize >= 32) {
            sdata[tid] = sdata[tid] > sdata[tid + 16] ? sdata[tid] : sdata[tid + 16];
        }
        if (blockSize >= 16) {
            sdata[tid] = sdata[tid] > sdata[tid + 8] ? sdata[tid] : sdata[tid + 8];
        }
        if (blockSize >= 8) {
            sdata[tid] = sdata[tid] > sdata[tid + 4] ? sdata[tid] : sdata[tid + 4];
        }
        if (blockSize >= 4) {
            sdata[tid] = sdata[tid] > sdata[tid + 2] ? sdata[tid] : sdata[tid + 2];
        }
        if (blockSize >= 2) {
            sdata[tid] = sdata[tid] > sdata[tid + 1] ? sdata[tid] : sdata[tid + 1];
        }
    }

    if (tid == 0) dst[blockIdx.x] = sdata[0];

    if (tid < 32) printf("returned %d\n", blockIdx.x);
    return;
}

__global__ void writeToImageKernel(float* weights, unsigned char* color, int num_pixels,
    int max_weight, const heatmap_colorscheme_t* colorscheme)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    for (int i = idx; i < num_pixels; i += blockDim.x * gridDim.x) {
        float val = weights[i] / (float)max_weight;
        size_t color_idx = (size_t)((float)(colorscheme->ncolors-1)*val + 0.5f);
        color[4*i] = (colorscheme->colors)[color_idx*4];
        color[4*i+1] = (colorscheme->colors)[color_idx*4+1];
        color[4*i+2] = (colorscheme->colors)[color_idx*4+2];
        color[4*i+3] = (colorscheme->colors)[color_idx*4+3];
    }
}

void cudaInit()
{
    hipMalloc(&pixel_weights, renderH * renderW * sizeof(float));
    hipMalloc(&pixel_color, renderH * renderW * sizeof(unsigned char));
    //hipMalloc(&max_buf, 1 * sizeof(float));
    //hipMalloc(&sizes, 2 * sizeof(int));

    hipMemcpy((void *)pixel_weights, (void *)hm->buf,
        renderH * renderW * sizeof(float), hipMemcpyHostToDevice);
}

__global__ void tempMax(float* src, float* dst, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    float &max_weight = dst[0];
    if (idx == 0) {
        for (int i = 0; i < n; i++) {
            max_weight = max_weight > src[i] ? max_weight : src[i];
        }
    }
}

void renderNewPointsCUDA(float x0, float y0, float w, float h, std::string filename)
{
    start_cuda = std::clock();
    float pt_width = w * 9 / renderW;
    float pt_height = h * 9 / renderH;
    //renderNewPointsKernel<<<128, 128>>>(x0, y0, w, h, renderW, renderH,
    //    pixel_weights, nodes, pt_width, pt_height);

    // get the maximum value of all weigths
    float max_weight;
    //tempMax<<<1, 1>>>(pixel_weights, max_buf, renderH * renderW);
    //hipMemcpy((void *)&max_weight, (void *)max_buf, 1 * sizeof(float), hipMemcpyDeviceToHost);

    hipMalloc(&max_buf, 1 * sizeof(float));

    int npixel = renderH * renderW;
    reduceMaxKernel<<<1, 512, 512 * sizeof(float)>>>(pixel_weights, max_buf, npixel);
    hipMemcpy((void *)&max_weight, (void *)max_buf, 1 * sizeof(float), hipMemcpyDeviceToHost);

    writeToImageKernel<<<128, 128>>>(pixel_weights, pixel_color, npixel, max_weight, heatmap_cs_default);
    hipDeviceSynchronize();
    std::cout << (std::clock() - start_cuda) * 1000  / (double) CLOCKS_PER_SEC << " ms\n";
    hipMemcpy((void *)ppmOutput->data, (void *)pixel_color,
        npixel * sizeof(unsigned char), hipMemcpyDeviceToHost);
    writePPMImage(ppmOutput, filename);
}