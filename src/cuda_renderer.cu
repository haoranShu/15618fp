#include "hip/hip_runtime.h"
#include <string>
#include <ctime>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_renderer.h"
#include "cdpQuadtree.h"

/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 #include <thrust/random.h>
 #include <thrust/device_vector.h>
 #include <hip/hip_runtime_api.h>
 
 #include "cdpQuadtree.h"
 
 class Points
 {
         float *m_x;
         float *m_y;
 
     public:
         // Constructor.
         __host__ __device__ Points() : m_x(NULL), m_y(NULL) {}
 
         // Constructor.
         __host__ __device__ Points(float *x, float *y) : m_x(x), m_y(y) {}
 
         // Get a point.
         __host__ __device__ __forceinline__ float2 get_point(int idx) const
         {
             return make_float2(m_x[idx], m_y[idx]);
         }
 
         // Set a point.
         __host__ __device__ __forceinline__ void set_point(int idx, const float2 &p)
         {
             m_x[idx] = p.x;
             m_y[idx] = p.y;
         }
 
         // Set the pointers.
         __host__ __device__ __forceinline__ void set(float *x, float *y)
         {
             m_x = x;
             m_y = y;
         }
 };
 
 ////////////////////////////////////////////////////////////////////////////////
 // A 2D bounding box
 ////////////////////////////////////////////////////////////////////////////////
 class Bounding_box
 {
         // Extreme points of the bounding box.
         float2 m_p_min;
         float2 m_p_max;
 
     public:
         // Constructor. Create a unit box.
         __host__ __device__ Bounding_box()
         {
             m_p_min = make_float2(0.0f, 0.0f);
             m_p_max = make_float2(1.0f, 1.0f);
         }
 
         // Compute the center of the bounding-box.
         __host__ __device__ void compute_center(float2 &center) const
         {
             center.x = 0.5f * (m_p_min.x + m_p_max.x);
             center.y = 0.5f * (m_p_min.y + m_p_max.y);
         }
 
         // The points of the box.
         __host__ __device__ __forceinline__ const float2 &get_max() const
         {
             return m_p_max;
         }
 
         __host__ __device__ __forceinline__ const float2 &get_min() const
         {
             return m_p_min;
         }

         __host__ __device__ bool overlaps(Bounding_box another_box)
         {
             float2 p3 = make_float2(another_box.m_p_min.x, another_box.m_p_max.y);
             float2 p4 = make_float2(another_box.m_p_max.x, another_box.m_p_min.y);
             return (contains(another_box.m_p_min) ||
                contains(another_box.m_p_max) ||
                contains(p3) || contains(p4));
         }

         // Does a box contain a point.
         __host__ __device__ bool contains(const float2 &p) const
         {
             return p.x >= m_p_min.x && p.x < m_p_max.x && p.y >= m_p_min.y && p.y < m_p_max.y;
         }
 
         // Define the bounding box.
         __host__ __device__ void set(float min_x, float min_y, float max_x, float max_y)
         {
             m_p_min.x = min_x;
             m_p_min.y = min_y;
             m_p_max.x = max_x;
             m_p_max.y = max_y;
         }
 };
 
 ////////////////////////////////////////////////////////////////////////////////
 // A node of a quadree.
 ////////////////////////////////////////////////////////////////////////////////
 class Quadtree_node
 {
         // The identifier of the node.
         int m_id;
         // The bounding box of the tree.
         Bounding_box m_bounding_box;
         // The range of points.
         int m_begin, m_end;
 
 
     public:
         // Constructor.
         __host__ __device__ Quadtree_node() : m_id(0), m_begin(0), m_end(0)
         {}
 
         // The ID of a node at its level.
         __host__ __device__ int id() const
         {
             return m_id;
         }
 
         // The ID of a node at its level.
         __host__ __device__ void set_id(int new_id)
         {
             m_id = new_id;
         }
 
         // The bounding box.
         __host__ __device__ __forceinline__ const Bounding_box &bounding_box() const
         {
             return m_bounding_box;
         }
 
         // Set the bounding box.
         __host__ __device__ __forceinline__ void set_bounding_box(float min_x, float min_y, float max_x, float max_y)
         {
             m_bounding_box.set(min_x, min_y, max_x, max_y);
         }
 
         // The number of points in the tree.
         __host__ __device__ __forceinline__ int num_points() const
         {
             return m_end - m_begin;
         }
 
         // The range of points in the tree.
         __host__ __device__ __forceinline__ int points_begin() const
         {
             return m_begin;
         }
 
         __host__ __device__ __forceinline__ int points_end() const
         {
             return m_end;
         }
 
         // Define the range for that node.
         __host__ __device__ __forceinline__ void set_range(int begin, int end)
         {
             m_begin = begin;
             m_end = end;
         }
 };
 
 ////////////////////////////////////////////////////////////////////////////////
 // Algorithm parameters.
 ////////////////////////////////////////////////////////////////////////////////
 struct Parameters
 {
     // Choose the right set of points to use as in/out.
     int point_selector;
     // The number of nodes at a given level (2^k for level k).
     int num_nodes_at_this_level;
     // The recursion depth.
     int depth;
     // The max value for depth.
     const int max_depth;
     // The minimum number of points in a node to stop recursion.
     const int min_points_per_node;
 
     // Constructor set to default values.
     __host__ __device__ Parameters(int max_depth, int min_points_per_node) :
         point_selector(0),
         num_nodes_at_this_level(1),
         depth(0),
         max_depth(max_depth),
         min_points_per_node(min_points_per_node)
     {}
 
     // Copy constructor. Changes the values for next iteration.
     __host__ __device__ Parameters(const Parameters &params, bool) :
         point_selector((params.point_selector+1) % 2),
         num_nodes_at_this_level(4*params.num_nodes_at_this_level),
         depth(params.depth+1),
         max_depth(params.max_depth),
         min_points_per_node(params.min_points_per_node)
     {}
 };
 
 ////////////////////////////////////////////////////////////////////////////////
 // Build a quadtree on the GPU. Use CUDA Dynamic Parallelism.
 //
 // The algorithm works as follows. The host (CPU) launches one block of
 // NUM_THREADS_PER_BLOCK threads. That block will do the following steps:
 //
 // 1- Check the number of points and its depth.
 //
 // We impose a maximum depth to the tree and a minimum number of points per
 // node. If the maximum depth is exceeded or the minimum number of points is
 // reached. The threads in the block exit.
 //
 // Before exiting, they perform a buffer swap if it is needed. Indeed, the
 // algorithm uses two buffers to permute the points and make sure they are
 // properly distributed in the quadtree. By design we want all points to be
 // in the first buffer of points at the end of the algorithm. It is the reason
 // why we may have to swap the buffer before leavin (if the points are in the
 // 2nd buffer).
 //
 // 2- Count the number of points in each child.
 //
 // If the depth is not too high and the number of points is sufficient, the
 // block has to dispatch the points into four geometrical buckets: Its
 // children. For that purpose, we compute the center of the bounding box and
 // count the number of points in each quadrant.
 //
 // The set of points is divided into sections. Each section is given to a
 // warp of threads (32 threads). Warps use __ballot and __popc intrinsics
 // to count the points. See the Programming Guide for more information about
 // those functions.
 //
 // 3- Scan the warps' results to know the "global" numbers.
 //
 // Warps work independently from each other. At the end, each warp knows the
 // number of points in its section. To know the numbers for the block, the
 // block has to run a scan/reduce at the block level. It's a traditional
 // approach. The implementation in that sample is not as optimized as what
 // could be found in fast radix sorts, for example, but it relies on the same
 // idea.
 //
 // 4- Move points.
 //
 // Now that the block knows how many points go in each of its 4 children, it
 // remains to dispatch the points. It is straightforward.
 //
 // 5- Launch new blocks.
 //
 // The block launches four new blocks: One per children. Each of the four blocks
 // will apply the same algorithm.
 ////////////////////////////////////////////////////////////////////////////////
 template< int NUM_THREADS_PER_BLOCK >
 __global__
 void build_quadtree_kernel(Quadtree_node *nodes, Points *points, Parameters params)
 {
     // The number of warps in a block.
     const int NUM_WARPS_PER_BLOCK = NUM_THREADS_PER_BLOCK / warpSize;
 
     // Shared memory to store the number of points.
     extern __shared__ int smem[];
 
     // s_num_pts[4][NUM_WARPS_PER_BLOCK];
     // Addresses of shared memory.
     volatile int *s_num_pts[4];
 
     for (int i = 0 ; i < 4 ; ++i)
         s_num_pts[i] = (volatile int *) &smem[i*NUM_WARPS_PER_BLOCK];
 
     // Compute the coordinates of the threads in the block.
     const int warp_id = threadIdx.x / warpSize;
     const int lane_id = threadIdx.x % warpSize;
 
     // Mask for compaction.
     int lane_mask_lt = (1 << lane_id) - 1; // Same as: asm( "mov.u32 %0, %%lanemask_lt;" : "=r"(lane_mask_lt) );
 
     // The current node.
     Quadtree_node &node = nodes[blockIdx.x];
 
     // The number of points in the node.
     int num_points = node.num_points();
 
     //
     // 1- Check the number of points and its depth.
     //
 
     // Stop the recursion here. Make sure points[0] contains all the points.
     if (params.depth >= params.max_depth || num_points <= params.min_points_per_node)
     {
         if (params.point_selector == 1)
         {
             int it = node.points_begin(), end = node.points_end();
 
             for (it += threadIdx.x ; it < end ; it += NUM_THREADS_PER_BLOCK)
                 if (it < end)
                     points[0].set_point(it, points[1].get_point(it));
         }
 
         return;
     }
 
     // Compute the center of the bounding box of the points.
     const Bounding_box &bbox = node.bounding_box();
     float2 center;
     bbox.compute_center(center);
 
     // Find how many points to give to each warp.
     int num_points_per_warp = max(warpSize, (num_points + NUM_WARPS_PER_BLOCK-1) / NUM_WARPS_PER_BLOCK);
 
     // Each warp of threads will compute the number of points to move to each quadrant.
     int range_begin = node.points_begin() + warp_id * num_points_per_warp;
     int range_end   = min(range_begin + num_points_per_warp, node.points_end());
 
     //
     // 2- Count the number of points in each child.
     //
 
     // Reset the counts of points per child.
     if (lane_id == 0)
     {
         s_num_pts[0][warp_id] = 0;
         s_num_pts[1][warp_id] = 0;
         s_num_pts[2][warp_id] = 0;
         s_num_pts[3][warp_id] = 0;
     }
 
     // Input points.
     const Points &in_points = points[params.point_selector];
 
     // Compute the number of points.
     for (int range_it = range_begin + lane_id ; __any(range_it < range_end) ; range_it += warpSize)
     {
         // Is it still an active thread?
         bool is_active = range_it < range_end;
 
         // Load the coordinates of the point.
         float2 p = is_active ? in_points.get_point(range_it) : make_float2(0.0f, 0.0f);
 
         // Count top-left points.
         int num_pts = __popc(__ballot(is_active && p.x < center.x && p.y >= center.y));
 
         if (num_pts > 0 && lane_id == 0)
             s_num_pts[0][warp_id] += num_pts;
 
         // Count top-right points.
         num_pts = __popc(__ballot(is_active && p.x >= center.x && p.y >= center.y));
 
         if (num_pts > 0 && lane_id == 0)
             s_num_pts[1][warp_id] += num_pts;
 
         // Count bottom-left points.
         num_pts = __popc(__ballot(is_active && p.x < center.x && p.y < center.y));
 
         if (num_pts > 0 && lane_id == 0)
             s_num_pts[2][warp_id] += num_pts;
 
         // Count bottom-right points.
         num_pts = __popc(__ballot(is_active && p.x >= center.x && p.y < center.y));
 
         if (num_pts > 0 && lane_id == 0)
             s_num_pts[3][warp_id] += num_pts;
     }
 
     // Make sure warps have finished counting.
     __syncthreads();
 
     //
     // 3- Scan the warps' results to know the "global" numbers.
     //
 
     // First 4 warps scan the numbers of points per child (inclusive scan).
     if (warp_id < 4)
     {
         int num_pts = lane_id < NUM_WARPS_PER_BLOCK ? s_num_pts[warp_id][lane_id] : 0;
 #pragma unroll
 
         for (int offset = 1 ; offset < NUM_WARPS_PER_BLOCK ; offset *= 2)
         {
             int n = __shfl_up(num_pts, offset, NUM_WARPS_PER_BLOCK);
 
             if (lane_id >= offset)
                 num_pts += n;
         }
 
         if (lane_id < NUM_WARPS_PER_BLOCK)
             s_num_pts[warp_id][lane_id] = num_pts;
     }
 
     __syncthreads();
 
     // Compute global offsets.
     if (warp_id == 0)
     {
         int sum = s_num_pts[0][NUM_WARPS_PER_BLOCK-1];
 
         for (int row = 1 ; row < 4 ; ++row)
         {
             int tmp = s_num_pts[row][NUM_WARPS_PER_BLOCK-1];
 
             if (lane_id < NUM_WARPS_PER_BLOCK)
                 s_num_pts[row][lane_id] += sum;
 
             sum += tmp;
         }
     }
 
     __syncthreads();
 
     // Make the scan exclusive.
     if (threadIdx.x < 4*NUM_WARPS_PER_BLOCK)
     {
         int val = threadIdx.x == 0 ? 0 : smem[threadIdx.x-1];
         val += node.points_begin();
         smem[threadIdx.x] = val;
     }
 
     __syncthreads();
 
     //
     // 4- Move points.
     //
 
     // Output points.
     Points &out_points = points[(params.point_selector+1) % 2];
 
     // Reorder points.
     for (int range_it = range_begin + lane_id ; __any(range_it < range_end) ; range_it += warpSize)
     {
         // Is it still an active thread?
         bool is_active = range_it < range_end;
 
         // Load the coordinates of the point.
         float2 p = is_active ? in_points.get_point(range_it) : make_float2(0.0f, 0.0f);
 
         // Count top-left points.
         bool pred = is_active && p.x < center.x && p.y >= center.y;
         int vote = __ballot(pred);
         int dest = s_num_pts[0][warp_id] + __popc(vote & lane_mask_lt);
 
         if (pred)
             out_points.set_point(dest, p);
 
         if (lane_id == 0)
             s_num_pts[0][warp_id] += __popc(vote);
 
         // Count top-right points.
         pred = is_active && p.x >= center.x && p.y >= center.y;
         vote = __ballot(pred);
         dest = s_num_pts[1][warp_id] + __popc(vote & lane_mask_lt);
 
         if (pred)
             out_points.set_point(dest, p);
 
         if (lane_id == 0)
             s_num_pts[1][warp_id] += __popc(vote);
 
         // Count bottom-left points.
         pred = is_active && p.x < center.x && p.y < center.y;
         vote = __ballot(pred);
         dest = s_num_pts[2][warp_id] + __popc(vote & lane_mask_lt);
 
         if (pred)
             out_points.set_point(dest, p);
 
         if (lane_id == 0)
             s_num_pts[2][warp_id] += __popc(vote);
 
         // Count bottom-right points.
         pred = is_active && p.x >= center.x && p.y < center.y;
         vote = __ballot(pred);
         dest = s_num_pts[3][warp_id] + __popc(vote & lane_mask_lt);
 
         if (pred)
             out_points.set_point(dest, p);
 
         if (lane_id == 0)
             s_num_pts[3][warp_id] += __popc(vote);
     }
 
     __syncthreads();
 
     //
     // 5- Launch new blocks.
     //
 
     // The last thread launches new blocks.
     if (threadIdx.x == NUM_THREADS_PER_BLOCK-1)
     {
         // The children.
         Quadtree_node *children = &nodes[params.num_nodes_at_this_level - (node.id() & ~3)];
 
         // The offsets of the children at their level.
         int child_offset = 4*node.id();
 
         // Set IDs.
         children[child_offset+0].set_id(4*node.id() + 0);
         children[child_offset+1].set_id(4*node.id() + 1);
         children[child_offset+2].set_id(4*node.id() + 2);
         children[child_offset+3].set_id(4*node.id() + 3);
 
         // Points of the bounding-box.
         const float2 &p_min = bbox.get_min();
         const float2 &p_max = bbox.get_max();
 
         // Set the bounding boxes of the children.
         children[child_offset+0].set_bounding_box(p_min.x , center.y, center.x, p_max.y);    // Top-left.
         children[child_offset+1].set_bounding_box(center.x, center.y, p_max.x , p_max.y);    // Top-right.
         children[child_offset+2].set_bounding_box(p_min.x , p_min.y , center.x, center.y);   // Bottom-left.
         children[child_offset+3].set_bounding_box(center.x, p_min.y , p_max.x , center.y);   // Bottom-right.
 
         // Set the ranges of the children.
         children[child_offset+0].set_range(node.points_begin(),   s_num_pts[0][warp_id]);
         children[child_offset+1].set_range(s_num_pts[0][warp_id], s_num_pts[1][warp_id]);
         children[child_offset+2].set_range(s_num_pts[1][warp_id], s_num_pts[2][warp_id]);
         children[child_offset+3].set_range(s_num_pts[2][warp_id], s_num_pts[3][warp_id]);
 
         // Launch 4 children.
         build_quadtree_kernel<NUM_THREADS_PER_BLOCK><<<4, NUM_THREADS_PER_BLOCK, 4 *NUM_WARPS_PER_BLOCK *sizeof(int)>>>(&children[child_offset], points, Parameters(params, true));
     }
 }
 
 ////////////////////////////////////////////////////////////////////////////////
 // Make sure a Quadtree is properly defined.
 ////////////////////////////////////////////////////////////////////////////////
 bool check_quadtree(const Quadtree_node *nodes, int idx, int num_pts, Points *pts, Parameters params)
 {
     const Quadtree_node &node = nodes[idx];
     int num_points = node.num_points();
     const Bounding_box &bbox = node.bounding_box();
 
     for (int it = node.points_begin() ; it < node.points_end() ; ++it)
     {
         if (it >= num_pts)
             return false;
 
         float2 p = pts->get_point(it);
 
         if (!bbox.contains(p))
             return false;
     }
 
     if (!(params.depth == params.max_depth || num_points <= params.min_points_per_node))
     {
         int sum = 0;
         for (int i = 0; i < 4; i++) {
             sum += nodes[4 * idx + params.num_nodes_at_this_level + i].num_points();
         }
 
         if (sum != num_points) {
             printf("[%d] node supposed to have %d points but children have %d\n", params.depth, num_points, sum);
         }
         return check_quadtree(&nodes[params.num_nodes_at_this_level], 4*idx+0, num_pts, pts, Parameters(params, true)) &&
                check_quadtree(&nodes[params.num_nodes_at_this_level], 4*idx+1, num_pts, pts, Parameters(params, true)) &&
                check_quadtree(&nodes[params.num_nodes_at_this_level], 4*idx+2, num_pts, pts, Parameters(params, true)) &&
                check_quadtree(&nodes[params.num_nodes_at_this_level], 4*idx+3, num_pts, pts, Parameters(params, true));
     }
 
     return true;
 }
 
 ////////////////////////////////////////////////////////////////////////////////
 // Allocate GPU structs, launch kernel and clean up
 ////////////////////////////////////////////////////////////////////////////////
 bool cdpQuadtree(float width, float height, float *xs, float *ys, float *ws, int num_points,
     Quadtree_node* nodes, Points* points)
 {
 
     // Find/set the device.
     // The test requires an architecture SM35 or greater (CDP capable).
     int cuda_device = findCudaDevice(1, NULL);
     hipDeviceProp_t deviceProps;
     checkCudaErrors(hipGetDeviceProperties(&deviceProps, cuda_device));
     int cdpCapable = (deviceProps.major == 3 && deviceProps.minor >= 5) || deviceProps.major >=4;
 
     printf("GPU device %s has compute capabilities (SM %d.%d)\n", deviceProps.name, deviceProps.major, deviceProps.minor);
 
     if (!cdpCapable)
     {
         std::cerr << "cdpQuadTree requires SM 3.5 or higher to use CUDA Dynamic Parallelism.  Exiting...\n" << std::endl;
         exit(EXIT_WAIVED);
     }
 
     int warp_size = deviceProps.warpSize;
 
     // Constants to control the algorithm.
     const int max_depth  = 12;
     const int min_points_per_node = 64;
 
     // Allocate memory for points.
     thrust::device_vector<float> x_d0(&xs[0], &xs[num_points]);
     thrust::device_vector<float> x_d1(num_points);
     thrust::device_vector<float> y_d0(&ys[0], &ys[num_points]);
     thrust::device_vector<float> y_d1(num_points);
 
     // Host structures to analyze the device ones.
     Points points_init[2];
     points_init[0].set(thrust::raw_pointer_cast(&x_d0[0]), thrust::raw_pointer_cast(&y_d0[0]));
     points_init[1].set(thrust::raw_pointer_cast(&x_d1[0]), thrust::raw_pointer_cast(&y_d1[0]));
 
     // Allocate memory to store points.
     //Points *points;
     checkCudaErrors(hipMalloc((void **) &points, 2*sizeof(Points)));
     checkCudaErrors(hipMemcpy(points, points_init, 2*sizeof(Points), hipMemcpyHostToDevice));
 
     // We could use a close form...
     int max_nodes = 0;
 
     for (int i = 0, num_nodes_at_level = 1 ; i < max_depth ; ++i, num_nodes_at_level *= 4)
         max_nodes += num_nodes_at_level;
 
     // Allocate memory to store the tree.
     Quadtree_node root;
     root.set_range(0, num_points);
     root.set_bounding_box(0, 0, width, height);
     //Quadtree_node *nodes;
     checkCudaErrors(hipMalloc((void **) &nodes, max_nodes*sizeof(Quadtree_node)));
     checkCudaErrors(hipMemcpy(nodes, &root, sizeof(Quadtree_node), hipMemcpyHostToDevice));
 
     // We set the recursion limit for CDP to max_depth.
     hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, max_depth);
 
     // Build the quadtree.
     Parameters params(max_depth, min_points_per_node);
     std::cout << "Launching CDP kernel to build the quadtree" << std::endl;
     const int NUM_THREADS_PER_BLOCK = 128; // Do not use less than 128 threads.
     const int NUM_WARPS_PER_BLOCK = NUM_THREADS_PER_BLOCK / warp_size;
     const size_t smem_size = 4*NUM_WARPS_PER_BLOCK*sizeof(int);
     build_quadtree_kernel<NUM_THREADS_PER_BLOCK><<<1, NUM_THREADS_PER_BLOCK, smem_size>>>(nodes, points, params);
     checkCudaErrors(hipGetLastError());
 
     /*
     // Copy points to CPU.
     thrust::host_vector<float> x_h(x_d0);
     thrust::host_vector<float> y_h(y_d0);
     Points host_points;
     host_points.set(thrust::raw_pointer_cast(&x_h[0]), thrust::raw_pointer_cast(&y_h[0]));
 
     // Copy nodes to CPU.
     Quadtree_node *host_nodes = new Quadtree_node[max_nodes];
     checkCudaErrors(hipMemcpy(host_nodes, nodes, max_nodes *sizeof(Quadtree_node), hipMemcpyDeviceToHost));
 
     // Validate the results.
     bool ok = check_quadtree(host_nodes, 0, num_points, &host_points, params);
     std::cout << "Results: " << (ok ? "OK" : "FAILED") << std::endl;
     
     // Free CPU memory.
     delete[] host_nodes;
 
     // Free memory.
     checkCudaErrors(hipFree(nodes));
     checkCudaErrors(hipFree(points));
     */
     return true;
 }
 
 ////////////////////////////////////////////////////////////////////////////////
 // Main entry point.
 ////////////////////////////////////////////////////////////////////////////////
 /*int main(int argc, char **argv)*/
 /*{*/
     /*// Find/set the device.*/
     /*// The test requires an architecture SM35 or greater (CDP capable).*/
     /*int cuda_device = findCudaDevice(argc, (const char **)argv);*/
     /*hipDeviceProp_t deviceProps;*/
     /*checkCudaErrors(hipGetDeviceProperties(&deviceProps, cuda_device));*/
     /*int cdpCapable = (deviceProps.major == 3 && deviceProps.minor >= 5) || deviceProps.major >=4;*/
 
     /*printf("GPU device %s has compute capabilities (SM %d.%d)\n", deviceProps.name, deviceProps.major, deviceProps.minor);*/
 
     /*if (!cdpCapable)*/
     /*{*/
         /*std::cerr << "cdpQuadTree requires SM 3.5 or higher to use CUDA Dynamic Parallelism.  Exiting...\n" << std::endl;*/
         /*exit(EXIT_WAIVED);*/
     /*}*/
 
     /*bool ok = cdpQuadtree(deviceProps.warpSize);*/
 
     /*return (ok ? EXIT_SUCCESS : EXIT_FAILURE);*/
 /*}*/


clock_t start_cuda;

__device__ void traverse(Quadtree_node *nodes, int idx, float *buf, Bounding_box box, 
    Points *pts, Parameters params, float pt_x, float pt_y, float x_reso, float y_reso,
    float* stamp)
{
    Quadtree_node* current = &nodes[idx];
    const Bounding_box &curr_box = current->bounding_box();
    if (!box.overlaps(curr_box)) {
        return;
    }

    printf("entered\n");
    int x_dist, y_dist;
    float2 p_min = curr_box.get_min();
    float2 p_max = curr_box.get_max();
    if (box.contains(p_max) && box.contains(p_min)) 
    {
        if (floor((p_min.x - pt_x + x_reso/2) / x_reso) ==
            floor((p_max.x - pt_x + x_reso/2) / x_reso) &&
            floor((p_min.y - pt_y + y_reso/2) / y_reso) ==
            floor((p_max.y - pt_y + y_reso/2) / y_reso)) {
            x_dist = (int)floor((p_min.x - pt_x + x_reso/2) / x_reso);
            y_dist = (int)floor((p_min.y - pt_y + y_reso/2) / y_reso);
            x_dist = x_dist > 4 ? 4 : x_dist;
            x_dist = x_dist < -4 ? -4 : x_dist;
            y_dist = y_dist > 4 ? 4 : y_dist;
            y_dist = y_dist < -4 ? -4 : y_dist;
            *buf = *buf + current->num_points() * stamp[9*(4 + y_dist) + (4 + x_dist)];
            printf("added\n");
        }
        return;
    }

    printf("entered2\n");
    if (params.depth == params.max_depth || current->num_points() <= params.min_points_per_node)
    {
        for (int it = current->points_begin() ; it < current->points_end() ; ++it)
        {
            float2 p = pts->get_point(it);
            if (box.contains(p)) {
                x_dist = (int)floor((p.x - pt_x + x_reso/2) / x_reso);
                y_dist = (int)floor((p.y - pt_y + y_reso/2) / y_reso); 
                *buf = *buf + stamp[9*(4 + y_dist) + (4 + x_dist)];
                printf("added\n");
            }
        }
        return;
    }
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+0, buf, box, pts, Parameters(params, true),
        pt_x, pt_y, x_reso, y_reso, stamp);
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+1, buf, box, pts, Parameters(params, true),
        pt_x, pt_y, x_reso, y_reso, stamp);
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+2, buf, box, pts, Parameters(params, true),
        pt_x, pt_y, x_reso, y_reso, stamp);
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+3, buf, box, pts, Parameters(params, true),
        pt_x, pt_y, x_reso, y_reso, stamp);
}

__global__ void renderNewPointsKernel(float x0, float y0, float w, float h, 
    int W, int H, float* buf, Quadtree_node* nodes, Points* points,
    float pt_width, float pt_height, float* stamp)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float x_reso = w / W;
    float y_reso = h / H;
    for (int i = idx; i < W * H; i += blockDim.x * gridDim.x) {
        buf[i] = 0;
        float pt_x = x0 + (i%W + 0.5) * x_reso;
        float pt_y = y0 + (i/W + 0.5) * y_reso;
        Bounding_box region;
        region.set(pt_x - pt_width/2, pt_y - pt_height/2,
            pt_x + pt_width/2, pt_y + pt_height/2);
        Parameters params(12, 64);
        traverse(nodes, 0, buf+i, region, points, params, pt_x, pt_y, x_reso, y_reso, stamp);
    }
}

__global__ void reduceMaxKernel(float* src, float* dst, int n)
{
    extern __shared__ float sdata[];

    int blockSize = blockDim.x;
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockSize * 2) + tid;
    int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;
    float temp = 0;

    while (i < n - blockSize) {
        temp = src[i] > src[i + blockSize] ? src[i] : src[i + blockSize];
        sdata[tid] = sdata[tid] > temp ? sdata[tid] : temp;
        i += gridSize;
    }
    while (i < n) {
        sdata[tid] = sdata[tid] > src[i] ? sdata[tid] : src[i]; 
    }
    __syncthreads();

    int startSize = 512;
    while (startSize > warpSize) {
        if (blockSize > startSize) {
            if (tid < startSize/2) { sdata[tid] = sdata[tid] > sdata[tid + startSize/2] ? sdata[tid] : sdata[tid + startSize/2]; }
            __syncthreads();
        }
        startSize /= 2;
    }

    // assuming a warpSize of 32
    if (tid < 32) {
        if (blockSize >= 64) {
            sdata[tid] = sdata[tid] > sdata[tid + 32] ? sdata[tid] : sdata[tid + 32];
        }
        if (blockSize >= 32) {
            sdata[tid] = sdata[tid] > sdata[tid + 16] ? sdata[tid] : sdata[tid + 16];
        }
        if (blockSize >= 16) {
            sdata[tid] = sdata[tid] > sdata[tid + 8] ? sdata[tid] : sdata[tid + 8];
        }
        if (blockSize >= 8) {
            sdata[tid] = sdata[tid] > sdata[tid + 4] ? sdata[tid] : sdata[tid + 4];
        }
        if (blockSize >= 4) {
            sdata[tid] = sdata[tid] > sdata[tid + 2] ? sdata[tid] : sdata[tid + 2];
        }
        if (blockSize >= 2) {
            sdata[tid] = sdata[tid] > sdata[tid + 1] ? sdata[tid] : sdata[tid + 1];
        }
    }

    if (tid == 0) dst[blockIdx.x] = sdata[0];

    return;
}

__global__ void writeToImageKernel(float* weights, unsigned char* color, int num_pixels,
    int max_weight, const heatmap_colorscheme_t* colorscheme)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    for (int i = idx; i < num_pixels; i += blockDim.x * gridDim.x) {
        float val = weights[i] / (float)max_weight;
        size_t color_idx = (size_t)((float)(colorscheme->ncolors-1)*val + 0.5f);
        color[4*i] = (colorscheme->colors)[color_idx*4];
        color[4*i+1] = (colorscheme->colors)[color_idx*4+1];
        color[4*i+2] = (colorscheme->colors)[color_idx*4+2];
        color[4*i+3] = (colorscheme->colors)[color_idx*4+3];
    }
}

void cudaInit()
{
    hipMalloc(&pixel_weights, renderH * renderW * sizeof(float));
    hipMalloc(&pixel_color, renderH * renderW * sizeof(unsigned char));
    hipMalloc(&cuda_stamp, 81 * sizeof(float));
    //hipMalloc(&max_buf, 1 * sizeof(float));
    //hipMalloc(&sizes, 2 * sizeof(int));

    hipMemcpy((void *)cuda_stamp, (void *)stamp,
        81 * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy((void *)pixel_weights, (void *)hm->buf,
    //    renderH * renderW * sizeof(float), hipMemcpyHostToDevice);
}

__global__ void tempMax(float* src, float* dst, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    float &max_weight = dst[0];
    if (idx == 0) {
        for (int i = 0; i < n; i++) {
            max_weight = max_weight > src[i] ? max_weight : src[i];
        }
    }
}

void renderNewPointsCUDA(float x0, float y0, float w, float h,
    std::string filename, float* stamp)
{
    printf("Here3\n");
    start_cuda = std::clock();
    float pt_width = w * 9 / renderW;
    float pt_height = h * 9 / renderH;

    printf("Here3\n");
    renderNewPointsKernel<<<128, 128>>>(x0, y0, w, h, renderW, renderH,
        pixel_weights, cuda_nodes, cuda_points, pt_width, pt_height, stamp);

    // get the maximum value of all weigths
    float max_weight;
    //tempMax<<<1, 1>>>(pixel_weights, max_buf, renderH * renderW);
    //hipMemcpy((void *)&max_weight, (void *)max_buf, 1 * sizeof(float), hipMemcpyDeviceToHost);

    printf("Here4\n");
    hipMalloc(&max_buf, 1 * sizeof(float));

    int npixel = renderH * renderW;
    reduceMaxKernel<<<1, 512, 512 * sizeof(float)>>>(pixel_weights, max_buf, npixel);
    hipMemcpy((void *)&max_weight, (void *)max_buf, 1 * sizeof(float), hipMemcpyDeviceToHost);

    writeToImageKernel<<<128, 128>>>(pixel_weights, pixel_color, npixel, max_weight, heatmap_cs_default);
    hipDeviceSynchronize();
    std::cout << (std::clock() - start_cuda) * 1000  / (double) CLOCKS_PER_SEC << " ms\n";
    hipMemcpy((void *)ppmOutput->data, (void *)pixel_color,
        npixel * sizeof(unsigned char), hipMemcpyDeviceToHost);
    writePPMImage(ppmOutput, filename);
}