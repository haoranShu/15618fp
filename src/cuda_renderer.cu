#include "hip/hip_runtime.h"
#include <string>
#include <ctime>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_renderer.h"
#include "cdpQuadtree.h"

clock_t start_cuda;

__device__ void traverse(Quadtree_node* nodes, float* weight,
    float pt_width, float pt_height, float pt_x, float pt_y)
{

}

__global__ void renderNewPointsKernel(float x0, float y0, float w, float h, 
    int W, int H, float* buf, Quadtree_node* nodes, float pt_width, float pt_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < W * H; i += blockDim.x * gridDim.x) {
        buf[i] = 0;
        float pt_x = x0 + (i%W + 0.5) * w / W;
        float pt_y = y0 + (i/W + 0.5) * h / H;
        traverse(nodes, buf+i, pt_width, pt_height, pt_x, pt_y);
    }
}

template <int blockSize>
__global__ void reduceMaxKernel(float* src, float* dst, int n)
{
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * (blockSize * 2) + tid;
    int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;

    while (i < n) {
        sdata[tid] = src[i] > src[i + blockSize] ? src[i] : src[i + blockSize];
        i += gridSize;
    }
    __syncthreads();

    int startSize = 512;
    while (startSize > warpSize) {
        if (blockSize > startSize) {
            if (tid < startSize/2) { sdata[tid] = src[tid] > src[tid + startSize/2] ? src[tid] : src[tid + startSize/2]; }
            __syncthreads();
        }
        startSize /= 2;
    }

    // assuming a warpSize of 32
    if (tid < 32) {
        if (blockSize >= 64) {
            sdata[tid] = src[tid] > src[tid + 32] ? src[tid] : src[tid + 32];
        }
        if (blockSize >= 32) {
            sdata[tid] = src[tid] > src[tid + 16] ? src[tid] : src[tid + 16];
        }
        if (blockSize >= 16) {
            sdata[tid] = src[tid] > src[tid + 8] ? src[tid] : src[tid + 8];
        }
        if (blockSize >= 8) {
            sdata[tid] = src[tid] > src[tid + 4] ? src[tid] : src[tid + 4];
        }
        if (blockSize >= 4) {
            sdata[tid] = src[tid] > src[tid + 2] ? src[tid] : src[tid + 2];
        }
        if (blockSize >= 2) {
            sdata[tid] = src[tid] > src[tid + 1] ? src[tid] : src[tid + 1];
        }
    }

    if (tid == 0) dst[blockIdx.x] = sdata[0];
    if (tid == 0) {
        float sum = 0;
        for (int l = 0; l < n; l++) {
            sum = sum > src[l] ? sum : src[l];
        }
    }
}

__global__ void writeToImageKernel(float* weights, unsigned char* color, int num_pixels,
    int max_weight, const heatmap_colorscheme_t* colorscheme)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    for (int i = idx; i < num_pixels; i += blockDim.x * gridDim.x) {
        float val = weights[i] / (float)max_weight;
        size_t color_idx = (size_t)((float)(colorscheme->ncolors-1)*val + 0.5f);
        color[4*i] = (colorscheme->colors)[color_idx*4];
        color[4*i+1] = (colorscheme->colors)[color_idx*4+1];
        color[4*i+2] = (colorscheme->colors)[color_idx*4+2];
        color[4*i+3] = (colorscheme->colors)[color_idx*4+3];
    }
}

void cudaInit()
{
    hipMalloc(&pixel_weights, renderH * renderW * sizeof(float));
    hipMalloc(&pixel_color, renderH * renderW * sizeof(unsigned char));

    hipMemcpy((void *)pixel_weights, (void *)hm->buf,
        renderH * renderW * sizeof(float), hipMemcpyHostToDevice);
}

void renderNewPointsCUDA(float x0, float y0, float w, float h, std::string filename)
{
    printf("here\n");
    start_cuda = std::clock();
    float pt_width = w * 9 / renderW;
    float pt_height = h * 9 / renderH;
    //renderNewPointsKernel<<<128, 128>>>(x0, y0, w, h, renderW, renderH,
    //    pixel_weights, nodes, pt_width, pt_height);

    // get the maximum value of all weigths
    float max_weight = 0;
    for (int i = 0; i < renderH * renderW; i++) {
        printf("%d\n", i);
        max_weight = max_weight > pixel_weights[i] ? max_weight : pixel_weights[i];
    }
    printf("here\n");
    writeToImageKernel<<<128, 128>>>(pixel_weights, pixel_color, renderH * renderW, max_weight, heatmap_cs_default);
    printf("here\n");
    hipDeviceSynchronize();
    std::cout << (std::clock() - start_cuda) * 1000  / (double) CLOCKS_PER_SEC << " ms\n";
    hipMemcpy((void *)ppmOutput->data, (void *)pixel_color,
        renderH * renderW * sizeof(unsigned char), hipMemcpyDeviceToHost);
    writePPMImage(ppmOutput, filename);
}