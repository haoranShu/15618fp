#include "hip/hip_runtime.h"
#include <string>
#include <ctime>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_renderer.h"
#include "cdpQuadtree.h"

clock_t start_cuda;

__device__ void traverse(Quadtree_node *nodes, int idx, float *buf, Bounding_box &box, 
    Points *pts, Parameters params, float pt_x, float pt_y, float x_reso, float y_reso,
    float* stamp)
{
    Quadtree_node current = nodes[idx];
    Bounding_box curr_box = current.bounding_box();
    if (!box.overlaps(curr_box))
        return;

    int x_dist, y_dist;
    if (box.contains(curr_box)) 
    {
        if ((floor)((curr_box.m_p_min.x - pt_x + x_reso/2) / x_reso) ==
            (floor)((curr_box.m_p_max.x - pt_x + x_reso/2) / x_reso) &&
            (floor)((curr_box.m_p_min.y - pt_y + y_reso/2) / y_reso) ==
            (floor)((curr_box.m_p_max.y - pt_y + y_reso/2) / y_reso)) {
            x_dist = (int)(floor)(curr_box.m_p_min.x - pt_x + x_reso/2) / x_reso);
            y_dist = (int)(floor)(curr_box.m_p_min.y - pt_y + y_reso/2) / y_reso);
            x_dist = x_dist > 4 ? 4 : x_dist;
            x_dist = x_dist < -4 ? -4 : x_dist;
            y_dist = y_dist > 4 ? 4 : y_dist;
            y_dist = y_dist < -4 ? -4 : y_dist;
            *buf = *buf + current.num_points() * stamp[9*(4 + y_dist) + (4 + x_dist)];
        }
        return;
    }

    if (params.depth == params.max_depth || current.num_points() <= params.min_points_per_node)
    {
        for (int it = node.points_begin() ; it < node.points_end() ; ++it)
        {
            float2 p = pts->get_point(it);
            if (box.contains(p)) {
                x_dist = (int)(floor)(p.x - pt_x + x_reso/2) / x_reso);
                y_dist = (int)(floor)(p.y - pt_y + y_reso/2) / y_reso); 
                *buf = *buf + stamp[9*(4 + y_dist) + (4 + x_dist)];
            }
        }
        return;
    }
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+0, buf, box, pts, Parameters(params, true),
        pt_x, pt_y, x_reso, y_reso, stamp);
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+1, buf, box, pts, Parameters(params, true),
        pt_x, pt_y, x_reso, y_reso, stamp);
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+2, buf, box, pts, Parameters(params, true),
        pt_x, pt_y, x_reso, y_reso, stamp);
    traverse(&nodes[params.num_nodes_at_this_level], 4*idx+3, buf, box, pts, Parameters(params, true),
        pt_x, pt_y, x_reso, y_reso, stamp);
}

__global__ void renderNewPointsKernel(float x0, float y0, float w, float h, 
    int W, int H, float* buf, Quadtree_node* nodes, Points* points,
    float pt_width, float pt_height, float* stamp)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float x_reso = w / W;
    float y_reso = h / H;
    for (int i = idx; i < W * H; i += blockDim.x * gridDim.x) {
        buf[i] = 0;
        float pt_x = x0 + (i%W + 0.5) * x_reso;
        float pt_y = y0 + (i/W + 0.5) * y_reso;
        Bounding_box region();
        region.set(pt_x - pt_width/2, pt_y - pt_height/2,
            pt_x + pt_width/2, pt_y + pt_height/2);
        Parameters params(12, 64);
        traverse(nodes, 0, buf+i, region, points, params, pt_x, pt_y, x_reso, y_reso, stamp);
    }
}

__global__ void reduceMaxKernel(float* src, float* dst, int n)
{
    extern __shared__ float sdata[];

    int blockSize = blockDim.x;
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockSize * 2) + tid;
    int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;
    float temp = 0;

    while (i < n - blockSize) {
        temp = src[i] > src[i + blockSize] ? src[i] : src[i + blockSize];
        sdata[tid] = sdata[tid] > temp ? sdata[tid] : temp;
        i += gridSize;
    }
    while (i < n) {
        sdata[tid] = sdata[tid] > src[i] ? sdata[tid] : src[i]; 
    }
    __syncthreads();

    int startSize = 512;
    while (startSize > warpSize) {
        if (blockSize > startSize) {
            if (tid < startSize/2) { sdata[tid] = sdata[tid] > sdata[tid + startSize/2] ? sdata[tid] : sdata[tid + startSize/2]; }
            __syncthreads();
        }
        startSize /= 2;
    }

    // assuming a warpSize of 32
    if (tid < 32) {
        if (blockSize >= 64) {
            sdata[tid] = sdata[tid] > sdata[tid + 32] ? sdata[tid] : sdata[tid + 32];
        }
        if (blockSize >= 32) {
            sdata[tid] = sdata[tid] > sdata[tid + 16] ? sdata[tid] : sdata[tid + 16];
        }
        if (blockSize >= 16) {
            sdata[tid] = sdata[tid] > sdata[tid + 8] ? sdata[tid] : sdata[tid + 8];
        }
        if (blockSize >= 8) {
            sdata[tid] = sdata[tid] > sdata[tid + 4] ? sdata[tid] : sdata[tid + 4];
        }
        if (blockSize >= 4) {
            sdata[tid] = sdata[tid] > sdata[tid + 2] ? sdata[tid] : sdata[tid + 2];
        }
        if (blockSize >= 2) {
            sdata[tid] = sdata[tid] > sdata[tid + 1] ? sdata[tid] : sdata[tid + 1];
        }
    }

    if (tid == 0) dst[blockIdx.x] = sdata[0];

    return;
}

__global__ void writeToImageKernel(float* weights, unsigned char* color, int num_pixels,
    int max_weight, const heatmap_colorscheme_t* colorscheme)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    for (int i = idx; i < num_pixels; i += blockDim.x * gridDim.x) {
        float val = weights[i] / (float)max_weight;
        size_t color_idx = (size_t)((float)(colorscheme->ncolors-1)*val + 0.5f);
        color[4*i] = (colorscheme->colors)[color_idx*4];
        color[4*i+1] = (colorscheme->colors)[color_idx*4+1];
        color[4*i+2] = (colorscheme->colors)[color_idx*4+2];
        color[4*i+3] = (colorscheme->colors)[color_idx*4+3];
    }
}

void cudaInit()
{
    hipMalloc(&pixel_weights, renderH * renderW * sizeof(float));
    hipMalloc(&pixel_color, renderH * renderW * sizeof(unsigned char));
    hipMalloc(&cuda_stamp, 81 * sizeof(float));
    //hipMalloc(&max_buf, 1 * sizeof(float));
    //hipMalloc(&sizes, 2 * sizeof(int));

    hipMemcpy((void *)cuda_stamp, (void *)stamp,
        81 * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy((void *)pixel_weights, (void *)hm->buf,
    //    renderH * renderW * sizeof(float), hipMemcpyHostToDevice);
}

__global__ void tempMax(float* src, float* dst, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    float &max_weight = dst[0];
    if (idx == 0) {
        for (int i = 0; i < n; i++) {
            max_weight = max_weight > src[i] ? max_weight : src[i];
        }
    }
}

void renderNewPointsCUDA(float x0, float y0, float w, float h,
    std::string filename, float* stamp)
{
    start_cuda = std::clock();
    float pt_width = w * 9 / renderW;
    float pt_height = h * 9 / renderH;

    renderNewPointsKernel<<<128, 128>>>(x0, y0, w, h, renderW, renderH,
        pixel_weights, cuda_nodes, cuda_points, pt_width, pt_height, cuda_stamp);

    // get the maximum value of all weigths
    float max_weight;
    //tempMax<<<1, 1>>>(pixel_weights, max_buf, renderH * renderW);
    //hipMemcpy((void *)&max_weight, (void *)max_buf, 1 * sizeof(float), hipMemcpyDeviceToHost);

    hipMalloc(&max_buf, 1 * sizeof(float));

    int npixel = renderH * renderW;
    reduceMaxKernel<<<1, 512, 512 * sizeof(float)>>>(pixel_weights, max_buf, npixel);
    hipMemcpy((void *)&max_weight, (void *)max_buf, 1 * sizeof(float), hipMemcpyDeviceToHost);

    writeToImageKernel<<<128, 128>>>(pixel_weights, pixel_color, npixel, max_weight, heatmap_cs_default);
    hipDeviceSynchronize();
    std::cout << (std::clock() - start_cuda) * 1000  / (double) CLOCKS_PER_SEC << " ms\n";
    hipMemcpy((void *)ppmOutput->data, (void *)pixel_color,
        npixel * sizeof(unsigned char), hipMemcpyDeviceToHost);
    writePPMImage(ppmOutput, filename);
}